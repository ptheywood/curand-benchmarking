#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <typeinfo>
#include <algorithm>

// Include the header for device random number generation.
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "CUDAErrorChecking.cuh"
#include "NVTXUtil.cuh"
#include "CUDAEventTimer.cuh"

#define USE_GRIDSTRIDE

/** 
 * Curand generator types
 * ## Psuedo RNG
 * HIPRAND_RNG_PSEUDO_XORWOW        hiprandStateXORWOW_t
 * HIPRAND_RNG_PSEUDO_MRG32K3A      hiprandStateMRG32k3a_t
 * HIPRAND_RNG_PSEUDO_MTGP32        hiprandStateMtgp32_t
 * HIPRAND_RNG_PSEUDO_PHILOX4_32_10 hiprandStatePhilox4_32_10_t
 * 
 * ## Host API only  
 * HIPRAND_RNG_PSEUDO_MT19937
 *
 * ## Quasi
 * HIPRAND_RNG_QUASI_SOBOL32           hiprandStateSobol32_t
 * HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32 hiprandStateScrambledSobol32_t
 * HIPRAND_RNG_QUASI_SOBOL64           hiprandStateSobol64_t
 * HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64 hiprandStateScrambledSobol64_t
 */

template<typename T>
__global__ void curand_initialise(const unsigned int STATES, const unsigned long long int seed, T* curandStates) {
    for (unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < STATES; idx += blockDim.x * gridDim.x) {
        hiprand_init(seed, idx, 0, &curandStates[idx]);
    }
}

// @todo - kernel which does memory accesses but doesn't use the kernel for overhead measurement.



// @note - risk of it being optimised out as not using the result. 
// @todo double variants
// @todo normal / lognormal
// @todo float2/float4/double2 variants.
template<typename T>
__global__ void curand_uniformf_sample(const unsigned int STATES, const unsigned int SAMPLES_PER_STATE, T* curandStates) {
    for (unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < STATES; idx += blockDim.x * gridDim.x) {
        // Load state into register.
        T state = curandStates[idx];
        float value = 0.f;
        for(unsigned int sampleIdx = 0; sampleIdx < SAMPLES_PER_STATE; sampleIdx++){
            value = hiprand_uniform(&state);
            // @todo make sure this isnt' being optimised out.
        }
        // Update the state in global memory.
        curandStates[idx] = state;
    }
}


template<typename T>
bool allocate(const unsigned int STATES, T ** d_curandStates, size_t * allocatedBytes){
    NVTX_RANGE("allocate");
    
    bool success = true;
    size_t bytes = STATES * sizeof(T);

    CUDA_CALL(hipMalloc((void**)d_curandStates, bytes));     
    *allocatedBytes = bytes;

    return success;
}

template<typename T>
void deallocate(T ** d_curandStates){
    CUDA_CALL(hipFree(*d_curandStates));
    *d_curandStates = nullptr;
}

template <typename T>
void curand_bench(const unsigned int STATES, const unsigned int SAMPLES_PER_STATE, const unsigned int REPS, const bool AGGREGATE_OUTPUT){
    NVTX_RANGE(typeid(T).name());

    // Get some information about the current device.
    int numSMs;
    int device;
    CUDA_CALL(hipGetDevice(&device));
    CUDA_CALL(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device));


    size_t totalCurandStateBytes = 0;
    double totalAllocMillis = 0.;
    double totalInitMillis = 0.;
    double totalUniformfSampleMillis = 0.;
    double totalDeallocMillis = 0.;


    // Repeat a few times to end up with better timings.
    for(unsigned int rep = 0; rep < REPS; rep++){
        // Prep timers
        CUDAEventTimer allocTimer = CUDAEventTimer();
        CUDAEventTimer initTimer = CUDAEventTimer();
        CUDAEventTimer uniformfSampleTimer = CUDAEventTimer();
        CUDAEventTimer deallocTimer = CUDAEventTimer();

        // prep occupancy
        int gridSize = 0;
        int minGridSize = 0;
        int blockSize = 0;

        // prep storage
        T * d_curandStates = nullptr; 
        size_t curandStateBytes = 0;

        // Prep the seed
        const unsigned long long int seed = rep;


        // Allocate hiprand (and time) 
        allocTimer.start();
        allocate(STATES, &d_curandStates, &curandStateBytes);
        allocTimer.stop();
        allocTimer.sync();

        // Initialise hiprand (and time)
        CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, curand_initialise<T>, 0, 0));
        gridSize = (STATES + blockSize - 1) / blockSize;
        #ifdef USE_GRIDSTRIDE
        gridSize = (std::min)(gridSize, minGridSize);
        #endif
        initTimer.start();
        // printf("Launching curand_initialise<T><%d,%d>\n", gridSize, blockSize);
        curand_initialise<T><<< gridSize, blockSize >>>(STATES, seed, d_curandStates); 
        initTimer.stop();
        initTimer.sync();


        // Sample from hiprand (and time)
        CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, curand_uniformf_sample<T>, 0, 0));
        gridSize = (STATES + blockSize - 1) / blockSize; 
        #ifdef USE_GRIDSTRIDE
        gridSize = (std::min)(gridSize, minGridSize);
        #endif

        uniformfSampleTimer.start();
        // printf("Launching curand_uniformf_sample<T><%d,%d>\n", gridSize, blockSize);
        curand_uniformf_sample<T><<< gridSize, blockSize >>>(STATES, SAMPLES_PER_STATE, d_curandStates); 
        uniformfSampleTimer.stop();
        uniformfSampleTimer.sync();


        // Free (dont' time)
        deallocTimer.start();
        deallocate(&d_curandStates);
        deallocTimer.stop();
        deallocTimer.sync();

        // Output data for this run. as csv.
        float allocMillis = allocTimer.getElapsedMilliseconds();
        float initMillis = initTimer.getElapsedMilliseconds();
        float uniformfSampleMillis = uniformfSampleTimer.getElapsedMilliseconds();
        float deallocMillis = deallocTimer.getElapsedMilliseconds();

        if (!AGGREGATE_OUTPUT) {
            // event timers have 0.5us resolution, so %.4f ms
            printf(
                "%s,%u,%u,%llu,%zu,%.4f,%.4f,%.4f,%.4f\n", 
                typeid(T).name(),
                STATES, 
                SAMPLES_PER_STATE,
                seed,
                curandStateBytes,
                allocMillis,
                initMillis,
                uniformfSampleMillis,
                deallocMillis
            );
        } else {
            totalCurandStateBytes += curandStateBytes;
            totalAllocMillis += allocMillis;
            totalInitMillis += initMillis;
            totalUniformfSampleMillis += uniformfSampleMillis;
            totalDeallocMillis += deallocMillis;
        }
    }
    if (AGGREGATE_OUTPUT) {
        // event timers have 0.5us resolution, so %.4f ms
        printf(
            "%s,%u,%u,%u,%f,%.4f,%.4f,%.4f,%.4f\n", 
            typeid(T).name(),
            STATES, 
            SAMPLES_PER_STATE,
            REPS,
            totalCurandStateBytes / (float)REPS,
            totalAllocMillis / (float)REPS,
            totalInitMillis / (float)REPS,
            totalUniformfSampleMillis / (float)REPS,
            totalDeallocMillis / (float)REPS
        );
    }
}

bool benchmark(const unsigned int STATES, const unsigned int SAMPLES_PER_STATE, const unsigned int REPS, const bool AGGREGATE_OUTPUT){
    // Push a range marker.
    NVTX_RANGE("benchmark");

    // print the header
    if(AGGREGATE_OUTPUT){
        printf("engine,threads,samples_per_thread,repetitions,mean_bytes,mean_alloc_ms,mean_init_ms,mean_uniformf_ms,mean_dealloc_ms\n");
    } else {
        printf("engine,threads,samples_per_thread,seed,bytes,alloc_ms,init_ms,uniformf_ms,dealloc_ms\n");
    }

    // Xorwow
    curand_bench<hiprandStateXORWOW_t>(STATES, SAMPLES_PER_STATE, REPS, AGGREGATE_OUTPUT);
    // MRG
    curand_bench<hiprandStateMRG32k3a_t>(STATES, SAMPLES_PER_STATE, REPS, AGGREGATE_OUTPUT);
    // MTG - requires special initialisation + a syncthreads per block, so not worth considering for our use-case.
    // curand_bench<hiprandStateMtgp32_t>(STATES, SAMPLES_PER_STATE, REPS, AGGREGATE_OUTPUT);
    // Philox
    curand_bench<hiprandStatePhilox4_32_10_t>(STATES, SAMPLES_PER_STATE, REPS, AGGREGATE_OUTPUT);

    return true;
}

void cudaInit(){
    NVTX_RANGE("cudaInit");
    // Free the nullptr to initialise the cuda context.
    CUDA_CALL(hipFree(0));
}

int main(int argc, char * argv[]){
    // Early initialise the cuda context to improve profiling clarity.
    cudaInit();

    // @todo - cli args.
    // @todo - better results reporting - performance is better than runtime.
    // Probably better to ask for N samples in total, and calc threads based on that (or use a grid strided loop + full device launch.)
    const unsigned int STATES = 262144;
    const unsigned int SAMPLES_PER_STATE = 65536;
    // const unsigned int SAMPLES_PER_STATE = 1048576;
    const unsigned int REPS = 5;
    const bool AGGREGATE_OUTPUT = true;

    // Run some stuff.
    bool success = benchmark(STATES, SAMPLES_PER_STATE, REPS, AGGREGATE_OUTPUT);

    // Reset the device.
    hipDeviceReset();

    return success ? EXIT_SUCCESS : EXIT_FAILURE;
}
